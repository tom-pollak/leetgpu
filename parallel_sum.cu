#include <stdlib.h>
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void parallel_sum_k(float* input, float* output, int N) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    // 1. Load current input
    float x = input[idx];
    __syncthreads();

    int max_depth = (int)__log2f(N);
    int max_scan = 1 << max_depth; // 2^max_depth
    for (int depth=0; depth < max_depth; depth++) {
        int stage_mod = 1 << depth; // 2^depth
        int prev_idx = idx - stage_mod;
        // pick this idx
        bool mask = prev_idx >= 0 && (idx + 1) % stage_mod == 0 && idx < max_scan;
        if (mask) x += input[prev_idx];
        __syncthreads();
        if (mask) input[idx] = x;
        __syncthreads();
    }

    // Process remainder

    if (idx == 0) {
        float final_scan_val = input[max_scan-1];
        for (int i=max_scan; i<N; i++) {
            final_scan_val += input[i];
        }
        *output = final_scan_val;
    }
}

// input, output are device pointers
void solve(const float* input, float* output, int N) {
    float* input_D;
    float* output_D;

    hipMalloc((void**) &input_D, N * sizeof(float));
    hipMemcpy(input_D, input, N * sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void**) &output_D, sizeof(float));

    parallel_sum_k<<<1, N>>>(input_D, output_D, N);

    hipMemcpy(output, output_D, sizeof(float), hipMemcpyDeviceToHost);
}

int main() {
    const int N = 1000;
    float input[N];
    float output = 0.0f;

    srand(42);
    for (int i = 0; i < N; i++) {
        input[i] = ((float)rand() / RAND_MAX) * 2000.0f - 1000.0f;
    }

    float gt = 0.0;
    for (int i=0; i<N; i++) gt += input[i];

    solve(input, &output, N);

    float diff = fabs(gt - output);
    printf("Ground truth: %f | Output: %f | Diff %f\n", gt, output, diff);
    return 0;
}
